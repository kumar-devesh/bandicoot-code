#include "hip/hip_runtime.h"
// Copyright 2023 Ryan Curtin (http://www.ratml.org/)
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
// ------------------------------------------------------------------------

// this kernel is technically incorrect if the size is not a factor of 2!
__global__
void
COOT_FN(PREFIX,sqrd_dbl_norm2_robust)(const eT1* in_mem,
                                      const UWORD n_elem,
                                      double* out_mem,
                                      const double max_val)
  {
  double* aux_mem = (double*) aux_shared_mem;

  const UWORD tid = threadIdx.x;
  UWORD i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
  const UWORD grid_size = blockDim.x * 2 * gridDim.x;

  aux_mem[tid] = 0;

  while (i + blockDim.x < n_elem)
    {
    // copy to local shared memory
    double v1 = ((double) in_mem[i]) / max_val;
    double v2 = ((double) in_mem[i + blockDim.x]) / max_val;
    aux_mem[tid] += (v1 * v1) + (v2 * v2);
    i += grid_size;
    }
  if (i < n_elem)
    {
    double v = ((double) in_mem[i]) / max_val;
    aux_mem[tid] += (v * v);
    }
  __syncthreads();

  for (UWORD s = blockDim.x / 2; s > 32; s >>= 1)
    {
    if (tid < s)
      {
      aux_mem[tid] += aux_mem[tid + s];
      }
    __syncthreads();
  }

  if (tid < 32) // unroll last warp's worth of work
    {
    // Since we are just accumulating now, we can use the accu() warp-reduce function.
    COOT_FN(PREFIX,accu_warp_reduce)(aux_mem, tid);
    }

  if (tid == 0)
    {
    out_mem[blockIdx.x] = aux_mem[0];
    }
  }
