#include "hip/hip_runtime.h"
// Copyright 2023 Ryan Curtin (http://www.ratml.org/)
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
// ------------------------------------------------------------------------

__global__
void
COOT_FN(PREFIX,var_colwise)(eT1* out,
                            const eT1* A,
                            const eT1* means,
                            const UWORD A_n_rows,
                            const UWORD A_n_cols,
                            const UWORD norm_correction)
  {
  const UWORD col = blockIdx.x * blockDim.x + threadIdx.x;
  if(col < A_n_cols)
    {
    const eT1* colptr = &(A[ col*A_n_rows ]);
    const eT1 mean_val = means[col];
    eT1 acc = (eT1) (0);
    for (UWORD i = 0; i < A_n_rows; ++i)
      {
      eT1 val = (colptr[i] - mean_val);
      acc += (val * val);
      }

    out[col] = (acc / (eT1) (A_n_rows - norm_correction));
    }
  }
