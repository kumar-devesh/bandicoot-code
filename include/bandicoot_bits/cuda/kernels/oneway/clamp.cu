#include "hip/hip_runtime.h"
// Copyright 2021 Marcus Edel (http://www.kurg.org/)
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
// ------------------------------------------------------------------------
__global__
void
COOT_FN(PREFIX,clamp)(const eT1* A_mem,
                      const eT1 min_val,
                      const eT1 max_val,
                      const UWORD num,
                      eT1* out_mem)
  {
  UWORD idx = blockIdx.x * blockDim.x + threadIdx.x;
  for(; idx < num; idx += blockDim.x * gridDim.x)
    {
    out_mem[idx] = max(min_val, min(max_val, A_mem[idx]));
    }
  }
