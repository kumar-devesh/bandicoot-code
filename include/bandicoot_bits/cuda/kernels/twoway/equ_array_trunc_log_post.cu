#include "hip/hip_runtime.h"
// Copyright 2022 Ryan Curtin (http://www.ratml.org/)
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
// ------------------------------------------------------------------------

__global__
void
COOT_FN(PREFIX,equ_array_trunc_log_post)(eT2* out,
                                         const eT1* A,
                                         const eT1 val_pre,
                                         const eT2 val_post,
                                         const UWORD N)
  {
  (void)(val_pre);
  (void)(val_post);
  const UWORD i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < N)
    {
    // To match Armadillo, we always use `double` as the intermediate type for any non-floating point type.
    const eT1 val = A[i];
    if (coot_is_fp(val))
      {
      const fp_eT1 fp_val = (fp_eT1) val;
      if (fp_val <= (fp_eT1) 0)
        {
        out[i] = (eT2) log(coot_type_min((fp_eT1) 0));
        }
      else if (isinf(fp_val))
        {
        out[i] = (eT2) log(coot_type_max((fp_eT1) 0));
        }
      else
        {
        out[i] = (eT2) ((eT1) log(fp_val));
        }
      }
    else
      {
      const double fp_val = (double) val;
      if (fp_val <= (fp_eT1) 0)
        {
        out[i] = (eT2) log(coot_type_min((double) 0));
        }
      else if (isinf(fp_val))
        {
        out[i] = (eT2) log(coot_type_max((double) 0));
        }
      else
        {
        out[i] = (eT2) ((eT1) log(fp_val));
        }
      }
    }
  }
