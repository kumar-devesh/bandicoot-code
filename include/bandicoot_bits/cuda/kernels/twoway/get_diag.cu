#include "hip/hip_runtime.h"
// Copyright 2019 Ryan Curtin (http://www.ratml.org/)
// 
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// http://www.apache.org/licenses/LICENSE-2.0
// 
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
// ------------------------------------------------------------------------

__global__
void
COOT_FN(PREFIX,get_diag)(eT2* out,
                         const eT1* A,
                         const UWORD n_rows,
                         const UWORD row_offset,
                         const UWORD col_offset,
                         const UWORD N)
  {
  const UWORD i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < N)
    {
    const UWORD index = (i + row_offset) + (i + col_offset)*n_rows;
    out[i] = (eT2) A[index];
    }
  }
