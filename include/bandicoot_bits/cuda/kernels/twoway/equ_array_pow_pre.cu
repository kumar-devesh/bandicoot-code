#include "hip/hip_runtime.h"
// Copyright 2022 Ryan Curtin (http://www.ratml.org/)
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
// ------------------------------------------------------------------------

__global__
void
COOT_FN(PREFIX,equ_array_pow_pre)(eT2* out,
                                   const eT1* A,
                                   const eT1 val_pre,
                                   const eT2 val_post,
                                   const UWORD N)
  {
  (void)(val_pre);
  (void)(val_post);
  const UWORD i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < N)
    {
    const fp_eT2 val = (fp_eT2) (eT2) A[i];
    out[i] = (eT2) pow(val, (fp_eT2) val_post);
    }
  }
